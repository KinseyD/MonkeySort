#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <hiprand/hiprand_kernel.h>

int array[] = { 16,15,14,13,12,11,10,9,8,7,6,5,4,3,2,1 };
const int arraySize = sizeof(array) / sizeof(int);

__device__ clock_t cost;

// 排序
__global__ void sortWithCuda(hiprandState* states, int** darray, size_t size)
{
	size_t idx = threadIdx.x + blockDim.x * blockIdx.x;
	unsigned x = hiprand(&states[idx]) % arraySize;
	unsigned y = hiprand(&states[idx]) % arraySize;
	// printf("(%u %u), ", x, y);
	if (x == y)
		return;
	darray[threadIdx.x][x] ^= darray[threadIdx.x][y];
	darray[threadIdx.x][y] ^= darray[threadIdx.x][x];
	darray[threadIdx.x][x] ^= darray[threadIdx.x][y];
	bool flag = true;
	for (int i = 1; i < arraySize; i++)
	{
		// printf("%d,", darray[threadIdx.x][i]);
		if (darray[threadIdx.x][i] < darray[threadIdx.x][i - 1])
			flag = false;
	}
	if (flag)
	{
		printf("线程(%u,%u)已完成，耗时%d\n", blockIdx.x, threadIdx.x, (clock() - cost) / CLK_TCK);
	}
}

// 设置开始时间
__global__ void setClock()
{
	cost = clock();
}

__global__ void randInit(hiprandState* states, size_t size)
{
	size_t idx = threadIdx.x + blockDim.x * blockIdx.x;
	hiprand_init(blockIdx.x, threadIdx.x, 0, &states[idx]);
}

__global__ void debug(int** array)
{
	for (size_t i = 0; i < arraySize; i++)
	{
		printf("%d ", array[32][i]);
	}
}

int main()
{
	size_t blk = 4e6;
	size_t trd = 32;

	int** deviceArray;
	int** midArray;
	hipMalloc((void***)&deviceArray, trd * sizeof(int*));
	midArray = (int**)malloc(trd * sizeof(int*));
	for (int i = 0; i < trd; i++)
	{
		hipMalloc((void**)&midArray[i], arraySize * sizeof(int));
		hipMemcpy(midArray[i], array, arraySize * sizeof(int), hipMemcpyHostToDevice);
	}
	hipMemcpy(deviceArray, midArray, trd * sizeof(int*), hipMemcpyHostToDevice);
	free(midArray);

	hiprandState* states;
	hipMalloc(&states, blk * trd * sizeof(hiprandState));

	randInit << <blk, trd >> > (states, blk * trd);
	hipDeviceSynchronize();

	printf("猴子开始工作！\n");
	setClock << <1, 1 >> > ();
	hipDeviceSynchronize();

	sortWithCuda << <blk, trd >> > (states, deviceArray, blk * trd);
	// hipDeviceSynchronize();

	return 0;
}