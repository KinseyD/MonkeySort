#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <hiprand/hiprand_kernel.h>

int array[] = { 22,21,20,19,18,17,16,15,14,13,12,11,10,9,8,7,6,5,4,3,2,1 };
const int arraySize = sizeof(array) / sizeof(int);

// 排序
__global__ void sortWithCuda(hiprandState* states, int** darray, bool* flag)
{
	size_t idx = threadIdx.x + blockDim.x * blockIdx.x;
	unsigned x = hiprand(&states[idx]) % arraySize;
	unsigned y = hiprand(&states[idx]) % arraySize;
	if (x == y)
		return;
	darray[threadIdx.x][x] ^= darray[threadIdx.x][y];
	darray[threadIdx.x][y] ^= darray[threadIdx.x][x];
	darray[threadIdx.x][x] ^= darray[threadIdx.x][y];
	for (int i = 1; i < arraySize; i++)
	{
		if (darray[threadIdx.x][i] < darray[threadIdx.x][i - 1])
			return;
	}
	printf("线程(%u,%u)已完成\n", blockIdx.x, threadIdx.x);
	*flag = true;
}

__global__ void randInit(hiprandState* states)
{
	size_t idx = threadIdx.x + blockDim.x * blockIdx.x;
	hiprand_init(blockIdx.x, threadIdx.x, 0, &states[idx]);
}

int main()
{
	size_t blk = 1e4;
	size_t trd = 32;

	// 在显存中开辟二维数组
	int** deviceArray;
	int** midArray;
	hipMalloc((void***)&deviceArray, trd * sizeof(int*));
	midArray = (int**)malloc(trd * sizeof(int*));
	for (int i = 0; i < trd; i++)
	{
		hipMalloc((void**)&midArray[i], arraySize * sizeof(int));
		hipMemcpy(midArray[i], array, arraySize * sizeof(int), hipMemcpyHostToDevice);
	}
	hipMemcpy(deviceArray, midArray, trd * sizeof(int*), hipMemcpyHostToDevice);
	free(midArray);

	// 初始化随机函数
	hiprandState* states;
	hipMalloc(&states, blk * trd * sizeof(hiprandState));
	randInit << <blk, trd >> > (states);
	hipDeviceSynchronize();

	// 记录是否完成
	bool* dFlag = nullptr;
	hipMalloc(&dFlag, sizeof(bool));
	bool hFlag = false;
	hipMemcpy(dFlag, &hFlag, sizeof(bool), hipMemcpyHostToDevice);

	printf("正在为%d个元素进行排序...\n\n", arraySize);

	// 准备计时
	hipEvent_t start, end;
	hipEventCreate(&start);
	hipEventCreate(&end);

	size_t rounds = 0;
	hipEventRecord(start);
	while (!hFlag)
	{
		sortWithCuda << <blk, trd >> > (states, deviceArray, dFlag);
		hipMemcpy(&hFlag, dFlag, sizeof(bool), hipMemcpyDeviceToHost);
		++rounds;
	}
	hipEventRecord(end);
	hipEventSynchronize(end);

	float cost;
	hipEventElapsedTime(&cost, start, end);
	if (cost > 1e4)
		printf("\n共有%llu个线程参与，耗时%.3fs\n", blk * trd * rounds, cost / 1000);
	else
		printf("\n共有%llu个线程参与，耗时%.2fms\n", blk * trd * rounds, cost);

	hipEventDestroy(start);
	hipEventDestroy(end);
	hipFree(deviceArray);
	hipFree(states);

	return 0;
}